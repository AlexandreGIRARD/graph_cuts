#include "hip/hip_runtime.h"
#include <iostream>

#include "gpu.hh"

#define HEIGHT_MAX 50
#define cudaCheckError() {                                                                \
        hipError_t e=hipGetLastError();                                                 \
        if(e!=hipSuccess) {                                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }                                                                                     \

__constant__ int x_nghb[4] = {0, 1, 0, -1}; // idx offset for x axis
__constant__ int y_nghb[4] = {-1, 0, 1, 0}; // idx offset for y axis
__constant__ int id_opp[4] = {2, 3, 0, 1};

__device__ inline int* at(int *addr, int x, int y, int pitch) {
    return (int*)((char*)addr + pitch * y + x * sizeof(int));
}

__global__ void relabel(int *excess, int *heights, int *tmp_heights, int *up,
        int *right, int *bottom, int *left, int width, int height, size_t pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    if (*at(excess, x, y, pitch) <= 0 || *at(heights, x, y, pitch) >= HEIGHT_MAX)
        return;
    int *ptr = at(tmp_heights, x, y, pitch);

    int *neighbors[4] = {up, right, bottom, left};
    int tmp_height = HEIGHT_MAX;
    for (int i = 0; i < 4; i++)
    {
        int new_x = x + x_nghb[i];
        int new_y = y + y_nghb[i];
        if (*at(neighbors[i], new_x, new_y, pitch) > 0)
            tmp_height = min(tmp_height, *at(heights, new_x, new_y, pitch) + 1);
    }
    *at(tmp_heights, x, y, pitch) = tmp_height;

}

__global__ void push(int *excess, int *heights, int *up, int *right,
        int *bottom, int *left, int width, int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;
    printf("x:%d|y:%d", x,y);
    int idx = y * width + x;
    if (excess[idx] <= 0 || heights[idx] >= HEIGHT_MAX)
        return;

    int *neighbors[4] = {up, right, bottom, left};

    for (auto i = 0; i < 4; i++) {
        auto idx_nghb = (y + y_nghb[i]) * width + (x + x_nghb[i]);
        if (y + y_nghb[i] < 0 || y + y_nghb[i] >= height ||
            x + x_nghb[i] < 0 || x + x_nghb[i] >= width)
            continue;
        if (heights[idx_nghb] != heights[idx] - 1)
            return;
        int flow = neighbors[i][idx] > excess[idx] ? excess[idx] : neighbors[i][idx];
        // make atomic here
        excess[idx] -= flow;
        excess[idx_nghb] += flow;

        neighbors[i][idx] -= flow;
        neighbors[id_opp[i]][idx_nghb] += flow;
    }
}

__global__ void print_value(int *array, int pitch, int width, int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;
    
}


int *duplicate_on_gpu(int *vect, int width, int height, size_t &pitch)
{
    int *array;

    hipMallocPitch(&array, &pitch, width * sizeof(int), height);
    cudaCheckError();
    hipMemcpy2D(array, pitch,
                 vect, width * sizeof(int),
                 width * sizeof(int), height, hipMemcpyHostToDevice);
    cudaCheckError();
    return array;
}

void max_flow_gpu(Graph graph)
{
    // Setting dimension
    int width  = graph._width;
    int height = graph._height;
    int w = std::ceil((float)width / 32);
    int h = std::ceil((float)height / 32);

    dim3 dimBlock(32, 32);
    dim3 dimGrid(w, h);

    auto arr = graph._heights;
    std::cout << std::endl<<arr[0] << " " << arr[1] <<" "<<arr[2]<<std::endl
        << arr[3] << " " << arr[4] <<" "<<arr[5]<<std::endl
        << arr[6] << " " << arr[7] <<" "<<arr[8]<<std::endl;

    // Allocate for gpu
    size_t pitch;
    int *excess = duplicate_on_gpu(graph._excess_flow, width, height, pitch);
    int *heights = duplicate_on_gpu(graph._heights, width, height, pitch);
    int *tmp_heights = duplicate_on_gpu(graph._heights, width, height, pitch);
    int *up = duplicate_on_gpu(graph._neighbors[0], width, height, pitch);
    int *right = duplicate_on_gpu(graph._neighbors[1], width, height, pitch);
    int *bottom = duplicate_on_gpu(graph._neighbors[2], width, height, pitch);
    int *left = duplicate_on_gpu(graph._neighbors[3], width, height, pitch);

    relabel<<<dimGrid, dimBlock>>>(excess, heights, tmp_heights, up, right, bottom, left,
            width, height, pitch);

    // try if memory was well duplicated
    std::cout << pitch << std::endl;
    int *new_arr = new int[9]();
    hipMemcpy2D(new_arr, width * sizeof(int),
                 tmp_heights, pitch,
                 width * sizeof(int), height, hipMemcpyDeviceToHost);
    arr = new_arr;
    std::cout << arr[0] << " " << arr[1] <<" "<<arr[2]<<std::endl
        << arr[3] << " " << arr[4] <<" "<<arr[5]<<std::endl
        << arr[6] << " " << arr[7] <<" "<<arr[8]<<std::endl;

   /* while ()
    {
        relabel<<<dimGrid, dimBlock>>>();
        push<<<dimGrid, dimBlock>>>();
    }*/
}
