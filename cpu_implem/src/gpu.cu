#include "hip/hip_runtime.h"
#include <iostream>

#include "gpu.hh"

#define HEIGHT_MAX 50
#define cudaCheckError() {                                                                \
        hipError_t e=hipGetLastError();                                                 \
        if(e!=hipSuccess) {                                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }                                                                                     \

__device__ inline int* at(int *addr, int x, int y, int pitch) {
    return (int*)(char*)addr + pitch * y + x * sizeof(int);
}

__global__ void relabel(int *excess, int *neighbors[4], int *heights, int width,
        int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    if (excess[idx] <= 0 || heights[idx] >= HEIGHT_MAX)
        return;
    const int x_nghb[4] = {0, 1, 0, -1}; // idx offset for x axis
    const int y_nghb[4] = {-1, 0, 1, 0}; // idx offset for y axis
    int tmp_height = HEIGHT_MAX;
    for (int i = 0; i < 4; i++)
    {
        int idx_nghb = (y + y_nghb[i]) * width + (x + x_nghb[i]);
        if (neighbors[i][idx_nghb] > 0)
            tmp_height = tmp_height > heights[idx_nghb] + 1 ? heights[idx_nghb] : tmp_height;
    }
    heights[idx] = tmp_height;

}

__global__ void push(int *excess, int *neighbors[4], int *heights, int width, 
        int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    if (excess[idx] <= 0 || heights[idx] >= HEIGHT_MAX)
        return;
    const int x_nghb[4] = {0, 1, 0, -1}; // idx offset for x axis
    const int y_nghb[4] = {-1, 0, 1, 0}; // idx offset for y axis
    const int id_opp[4] = {2, 3, 0, 1};

    for (auto i = 0; i < 4; i++) {
        auto idx_nghb = (y + y_nghb[i]) * width + (x + x_nghb[i]);
        if (y + y_nghb[i] < 0 || y + y_nghb[i] >= height ||
            x + x_nghb[i] < 0 || x + x_nghb[i] >= width)
            continue;
        if (heights[idx_nghb] != heights[idx] - 1)
            return;
        int flow = neighbors[i][idx] > excess[idx] ? excess[idx] : neighbors[i][idx];
        // make atomic here
        excess[idx] -= flow;
        excess[idx_nghb] += flow;

        neighbors[i][idx] -= flow;
        neighbors[id_opp[i]][idx_nghb] += flow;
    }
}

__global__ void print_value(int *array, int pitch, int width, int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;
    
}


int *duplicate_on_gpu(int *vect, int width, int height, size_t pitch)
{
    int *array;

    hipMallocPitch(&array, &pitch, width * sizeof(int), height);
    cudaCheckError();
    std::cout << "Pitch: "<<pitch<<std::endl;
    hipMemcpy2D(array, pitch,
                 vect, width * sizeof(int),
                 width * sizeof(int), height, hipMemcpyHostToDevice);
    cudaCheckError();
    return array;
}

void max_flow_gpu(Graph graph)
{
    // Setting dimension
    int width  = graph._width;
    int height = graph._height;
    int w = std::ceil((float)width / 32);
    int h = std::ceil((float)height / 32);

    dim3 dimBlock(32, 32);
    dim3 dimGrid(w, h);


    // Allocate for gpu
    size_t pitch;
    int *excess = duplicate_on_gpu(graph._excess_flow, width, height, pitch);
    int *heights = duplicate_on_gpu(graph._heights, width, height, pitch);
    int *tmp_heights;
    int *up = duplicate_on_gpu(graph._neighbors[0], width, height, pitch);
    int *right = duplicate_on_gpu(graph._neighbors[1], width, height, pitch);
    int *bottom = duplicate_on_gpu(graph._neighbors[2], width, height, pitch);
    int *left = duplicate_on_gpu(graph._neighbors[3], width, height, pitch);


   /* while ()
    {
        relabel<<<dimGrid, dimBlock>>>();
        push<<<dimGrid, dimBlock>>>();
    }*/
}
